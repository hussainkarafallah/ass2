

#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <iomanip>
#include <iostream>
#include <vector>
#include "hipblas.h"


hipblasHandle_t handle;
hipblasStatus_t stat = hipblasCreate(&handle);


__global__ void shit(
    const int M,
    const int N,
    float *d_A,
    float *d_X,
    float *d_Y
)
{
  
  for(int j = 0 ; j < N ; j++){
    d_Y[j] = 2.0;
  }
  printf("\n");
 
  printf("\n");
}

void initVec(const int N , float *vec , const float val){
  for(unsigned int i = 0 ; i < N ; i++)
    vec[i] = val;
}

void initMat(const int M , const int N , float *mat){
  const unsigned int COLUMN_STEP = 4;
  for(unsigned int row = 0 ; row < M ; row++){
    for(unsigned int col = 0 ; col < N ; col++){
      mat[col * M + row] = (col / COLUMN_STEP);
    }
  }
}

// Run the actual benchmark
void benchmark_matvec(const std::size_t M , const std::size_t N , const unsigned int n_repeat , int useCublas)
{

  float *h_A = (float*) malloc(M * N * sizeof(float));
  float *h_X = (float*) malloc(M * sizeof(float));
  float *h_Y = (float*) malloc(N * sizeof(float));
  
  // initialize vector to all ones
  initVec(M , h_X , 1);
  // initialize matrix so ith row has integer part of (i/100)
  // first 100 columns will be zeroes, second 100 columns will be ones ... etc
  initMat(M , N , h_A);
  // expected result of ith scalar of resulting vector is M * int(i/100)
  initVec(N , h_Y , 0);

  float *d_A , *d_X , *d_Y;

  hipMalloc(&d_X, M * sizeof(float));
  hipMalloc(&d_Y, N * sizeof(float));
  hipMalloc(&d_A, M * N * sizeof(float));
    
  hipMemcpy(d_X , h_X , M * sizeof(float) ,hipMemcpyHostToDevice);
  hipMemcpy(d_A , h_A , M * N * sizeof(float) ,hipMemcpyHostToDevice);

  
  std::vector<float> result_host(N);

  const unsigned int n_tests = 30;
  double best = 1e10, worst = 0, avg = 0;
  

  float alpha = 1.f , beta = 0.;

  for (unsigned int t = 0; t < n_tests; ++t)
    {
      // type of t1: std::chrono::steady_clock::time_point
      const auto t1 = std::chrono::steady_clock::now();

      for (unsigned int rep = 0; rep < n_repeat; ++rep){
        if(useCublas){
          stat =hipblasSgemv(handle, HIPBLAS_OP_N, M, N, &alpha, d_A, M, d_X, 1, &beta, d_Y, 1);
            if (stat != HIPBLAS_STATUS_SUCCESS){
              std::cout << "CUBLAS operation failed\n";
              std::abort();
            }
            
        }
        else{
          //const unsigned int n_blocks = (M + threads_per_block - 1) / threads_per_block;
          //dot_product<<<n_blocks, threads_per_block>>>(M , N , d_A , d_X ,d_Y);
          shit<<<1,1>>>(M , N , d_A , d_X , d_Y);
        }
      }

      hipDeviceSynchronize();
      // measure the time by taking the difference between the time point
      // before starting and now
      const double time =
        std::chrono::duration_cast<std::chrono::duration<double>>(
          std::chrono::steady_clock::now() - t1)
          .count();

      best  = std::min(best, (double) (time / n_repeat));
      worst = std::max(worst, (double) (time / n_repeat));
      avg += time / n_repeat;
    }

  // Copy the result back to the host
  hipMemcpy(result_host.data(), d_Y, N * sizeof(float), hipMemcpyDeviceToHost);
  
  for(int j = 0 ; j < N ; j++){
    printf("%.0lf " , result_host[j]);
  }

  int bad_result = 0;
  for(int i = 0 ; i < N ; i++){
    int expected = (i / 100) * M;
    if(result_host[i] != expected)
      bad_result = 1;
  }

  

  if (bad_result)
    std::cout << "Error in computation, some scalar in the vector is not as expected" << std::endl;

  // Free the memory on the device
  hipFree(d_A);
  hipFree(d_X);
  
  long long ops = 1ll * N * M;

  std::cout << "STREAM triad with "<< M << "rows and " << N <<" columns" 
            << std::setw(8) << 1e-6 * ops / best << " MUPD/s or " 
            << std::setw(8) << 1e-9 * sizeof(float) * ops / best << " GB/s" << std::endl;
}


int main(int argc, char **argv)
{

  if (stat != HIPBLAS_STATUS_SUCCESS){
    std::cout << "CUBLAS initialization failed\n";
    std::abort();
  }


  //printf("Plain CUDA:: \n");
  //benchmark_matvec(5000 , 5000 , 30, 0);
  
  printf("CUBLAS :: \n");
  for(int n = 20 ; n <= 20 ; n = (1 + n * 1.1)){
     benchmark_matvec(n , n , 1 , 0);
  }
  
  return 0;
}
