

#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <iomanip>
#include <iostream>
#include <vector>
#include "hipblas.h"

const unsigned int threads_per_block = 1024;

hipblasHandle_t handle;
hipblasStatus_t stat = hipblasCreate(&handle);

/*
__global__ void dot_product(
  const int M,
  const int N,
  const float *A,
  const float *X,
  float *Y
)
{
  const int row = threadIdx.x + blockIdx.x * blockDim.x;
  if(row >= M)
    return;
  float result = 0.0;
  for(unsigned int col = 0 ; col < N ; col++){
    result += A[col * M + row] * X[col];
  }

  Y[row] = result;
}*/

void initVec(const int N , float *vec , const float val){
  for(unsigned int i = 0 ; i < N ; i++)
    vec[i] = val;
}

void initMat(const int M , const int N , float *mat){
  for(unsigned int row = 0 ; row < M ; row++){
    for(unsigned int col = 0 ; col < N ; col++){
      mat[col * M + row] = (row / 100);
    }
  }
}

// Run the actual benchmark
void benchmark_triad(const std::size_t M , const std::size_t N , const unsigned int n_repeat , int useCublas)
{

  const float val = 97;

  float *h_A = (float*) malloc(M * N * sizeof(float));
  float *h_X = (float*) malloc(M * sizeof(float));
  float *h_Y = (float*) malloc(N * sizeof(float));
  
  // initialize vector to all ones
  initVec(M , h_X , 1);
  // initialize matrix so ith row has integer part of (i/100)
  // first 100 columns will be zeroes, second 100 columns will be ones ... etc
  initMat(M , N , h_A);
  // expected result of ith scalar of resulting vector is M * int(i/100)
  initVec(N , h_Y , 0);

  float *d_A , *d_X , *d_Y;

  hipMalloc(&d_X, M * sizeof(float));
  hipMalloc(&d_Y, N * sizeof(float));
  hipMalloc(&d_A, M * N * sizeof(float));
    
  hipMemcpy(d_X , h_X , M * sizeof(float) ,hipMemcpyHostToDevice);
  hipMemcpy(d_A , h_A , M * N * sizeof(float) ,hipMemcpyHostToDevice);

  
  std::vector<float> result_host(N);

  const unsigned int n_tests = 30;
  double best = 1e10, worst = 0, avg = 0;
  

  float alpha = 1.f , beta = 0.;

  for (unsigned int t = 0; t < n_tests; ++t)
    {
      // type of t1: std::chrono::steady_clock::time_point
      const auto t1 = std::chrono::steady_clock::now();

      for (unsigned int rep = 0; rep < n_repeat; ++rep){
        if(useCublas){
          stat =hipblasSgemv(handle, HIPBLAS_OP_T, M, N, &alpha, d_A, M, d_X, 1, &beta, d_Y, 1);
            if (stat != HIPBLAS_STATUS_SUCCESS){
              std::cout << "CUBLAS operation failed\n";
              std::abort();
            }
        }
        else{
          const unsigned int n_blocks = (M + threads_per_block - 1) / threads_per_block;
          //dot_product<<<n_blocks, threads_per_block>>>(M , N , d_A , d_X ,d_Y);
        }
      }

      hipDeviceSynchronize();
      // measure the time by taking the difference between the time point
      // before starting and now
      const double time =
        std::chrono::duration_cast<std::chrono::duration<double>>(
          std::chrono::steady_clock::now() - t1)
          .count();

      best  = std::min(best, (double) (time / n_repeat));
      worst = std::max(worst, (double) (time / n_repeat));
      avg += time / n_repeat;
    }

  // Copy the result back to the host
  hipMemcpy(result_host.data(), d_Y, N * sizeof(float), hipMemcpyDeviceToHost);
  
  int bad_result = 0;
  for(int i = 0 ; i < N ; i++){
    int expected = (i / 100) * M;
    if(result_host[i] != expected)
      bad_result = 1;
  }

  for(int i = 0 ; i < M ; i++){
    for(int j = 0 ; j < N ; j++){
      printf("%lf " , h_A[j * M + i]);
    }
    printf("\n");
  }
  for(int j = 0 ; j < N ; j++){
    printf("%lf " , h_X[j]);
  }
  printf("\n");

  if (bad_result)
    std::cout << "Error in computation, some scalar in the vector is not as expected" << std::endl;

  // Free the memory on the device
  hipFree(d_A);
  hipFree(d_X);
  
  long long ops = 1ll * N * M;

  std::cout << "STREAM triad with "<< M << "rows and " << N <<" columns" 
            << std::setw(8) << 1e-6 * ops / best << " MUPD/s or " 
            << std::setw(8) << 1e-9 * sizeof(float) * ops / best << " GB/s" << std::endl;
}


int main(int argc, char **argv)
{

  if (stat != HIPBLAS_STATUS_SUCCESS){
    std::cout << "CUBLAS initialization failed\n";
    std::abort();
  }


  //printf("Plain CUDA:: \n");
  //benchmark_triad(5000 , 5000 , 30, 0);
  
  printf("CUBLAS :: \n");
  for(int n = 32 ; n <= 32 ; n = (1 + n * 1.1)){
     benchmark_triad(n , n , 1 , 1);
  }
  
  return 0;
}
