

#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <iomanip>
#include <iostream>
#include <vector>
#include "hipblas.h"

const unsigned int threads_per_block = 128;

hipblasHandle_t handle;
hipblasStatus_t stat = hipblasCreate(&handle);


__global__ void dot_product(
  const int M,
  const int N,
  const float *A,
  const float *X,
  float *Y
)
{
  const int row = threadIdx.x + blockIdx.x * blockDim.x;
  if(row >= M)
    return;
  float result = 0.0;
  for(unsigned int col = 0 ; col < N ; col++){
    result += A[col * M + row] * X[col];
  }

  Y[row] = result;
}

void initVec(const int N , float *vec , const float val){
  for(unsigned int i = 0 ; i < N ; i++)
    vec[i] = val;
}

void initMat(const int M , const int N , float *mat , float val){
  for(unsigned int row = 0 ; row < M ; row++){
    for(unsigned int col = 0 ; col < N ; col++){
      mat[col * M + row] = val;
    }
  }
}

// Run the actual benchmark
void benchmark_triad(const std::size_t M , const std::size_t N , const unsigned int n_repeat , int useCublas)
{

  const float val = 97;

  float *h_A = (float*) malloc(M * N * sizeof(float));
  float *h_X = (float*) malloc(N * sizeof(float));
  float *h_Y = (float*) malloc(M * sizeof(float));
  
  initVec(N , h_X , 1);
  initVec(M , h_Y , 0);
  initMat(M , N , h_A , val);


  float *d_A , *d_X , *d_Y;
  // allocate matrix and vector
  
  hipMalloc(&d_X, N * sizeof(float));
  hipMalloc(&d_Y, M * sizeof(float));
  hipMalloc(&d_A, M * N * sizeof(float));
    
  hipMemcpy(d_X , h_X , N * sizeof(float) ,hipMemcpyHostToDevice);
  hipMemcpy(d_A , h_A , M * N * sizeof(float) ,hipMemcpyHostToDevice);

  
  const unsigned int n_blocks = (M + threads_per_block - 1) / threads_per_block;

  std::vector<float> result_host(M);

  const unsigned int n_tests = 30;
  double best = 1e10, worst = 0, avg = 0;
  

  float alpha = 1.f , beta = 0.;

  for (unsigned int t = 0; t < n_tests; ++t)
    {
      // type of t1: std::chrono::steady_clock::time_point
      const auto t1 = std::chrono::steady_clock::now();

      for (unsigned int rep = 0; rep < n_repeat; ++rep){
        if(useCublas){
          stat =hipblasSgemv(handle, HIPBLAS_OP_N, M, N, &alpha, d_A, M, d_X, 1, &beta, d_Y, 1);
            if (stat != HIPBLAS_STATUS_SUCCESS){
              std::cout << "CUBLAS operation failed\n";
              std::abort();
            }
        }
        else{
          dot_product<<<n_blocks, threads_per_block>>>(M , N , d_A , d_X ,d_Y);
        }
      }

      hipDeviceSynchronize();
      // measure the time by taking the difference between the time point
      // before starting and now
      const double time =
        std::chrono::duration_cast<std::chrono::duration<double>>(
          std::chrono::steady_clock::now() - t1)
          .count();

      best  = std::min(best, (double) (time / n_repeat));
      worst = std::max(worst, (double) (time / n_repeat));
      avg += time / n_repeat;
    }

  // Copy the result back to the host
  hipMemcpy(result_host.data(), d_Y, M * sizeof(float), hipMemcpyDeviceToHost);
  
  float targetResult = N * val;
  if (result_host[0] != targetResult)
    std::cout << "Error in computation, got "
              << result_host[0] << " instead of "<< targetResult
              << std::endl;

  // Free the memory on the device
  hipFree(d_A);
  hipFree(d_X);
  
  long long ops = 1ll * N * M;

  std::cout << "matrid vector multiplication with "<< M << " rows and " << N <<" columns" 
            << std::setw(8) << 1e-9 * sizeof(float) * ops / best << " GB/s" << std::endl;
}


int main(int argc, char **argv)
{

  if (stat != HIPBLAS_STATUS_SUCCESS){
    std::cout << "CUBLAS initialization failed\n";
    std::abort();
  }

  if (argc != 2){
    std::cout << "Error, add a single argument depending on experiment";
    std::abort();
  }

  long task = static_cast<long>(std::stod(argv[1]));
  // first task measure for n = m for simple parallelization and cublas
  if(task == 1){
    printf("Plain CUDA:: \n");
    for(int n = 100 ; n <= 10000 ; n = (1 + n * 1.1)){
      n = (n + 7) / 8 * 8;
      benchmark_triad(n , n , 10000 / n, 0);
    }
    printf("CUBLAS :: \n");
    for(int n = 100 ; n <= 10000 ; n = (1 + n * 1.1)){
      n = (n + 7) / 8 * 8;
      benchmark_triad(n , n , 10000 / n , 1);
    }
  }
  // measure for constant n for parallelization and cublas
  if(task == 2){

    printf("Plain CUDA:: \n");

    for(int m = 100 ; m <= 10000 ; m = (1 + m * 1.1)){
      m = (m + 7) / 8 * 8;
      benchmark_triad(m , 10000 , 10000 / m , 1);
    }

    printf("CUBLAS:: \n");

    for(int m = 100 ; m <= 10000 ; m = (1 + m * 1.1)){
      m = (m + 7) / 8 * 8;
      benchmark_triad(m , 10000 , 10000 / m , 1);
    }

  }
  // measure for constant m for parallelization and cublas
  if(task == 3){

    printf("Plain CUDA:: \n");

    for(int n = 100 ; n <= 10000 ; n = (1 + n * 1.1)){
      n = (n + 7) / 8 * 8;
      benchmark_triad(16384 , n , 10000 / n , 1);
    }

    printf("CUBLAS:: \n");
    
    for(int n = 100 ; n <= 10000 ; n = (1 + n * 1.1)){
      n = (n + 7) / 8 * 8;
      benchmark_triad(16384 , n , 10000 / n , 1);
    }

  }
  
  
  hipblasDestroy(handle);
  
  return 0;
}
