

#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <iomanip>
#include <iostream>
#include <vector>
#include "hipblas.h"

const unsigned int threads_per_block = 128;

hipblasHandle_t handle;
hipblasStatus_t stat = hipblasCreate(&handle);


__global__ void dot_product(
  const int M,
  const int N,
  const float *A,
  const float *X,
  float *Y
)
{
  const int row = threadIdx.x + blockIdx.x * blockDim.x;
  if(row >= M)
    return;
  float result = 0.0;
  for(unsigned int col = 0 ; col < N ; col++){
    result += A[col * M + row] * X[col];
  }

  Y[row] = result;
}

void initVec(const int N , float *vec , const float val){
  for(unsigned int i = 0 ; i < N ; i++)
    vec[i] = val;
}

void initMat(const int M , const int N , float *mat){
  for(unsigned int row = 0 ; row < M ; row++){
    for(unsigned int col = 0 ; col < N ; col++){
      mat[col * M + row] = col;
    }
  }
}

// Run the actual benchmark
void benchmark_triad(const std::size_t M , const std::size_t N , const int repeatBound, int useCublas)
{

  float *h_A = (float*) malloc(M * N * sizeof(float));
  float *h_X = (float*) malloc(N * sizeof(float));
  float *h_Y = (float*) malloc(M * sizeof(float));
  
  initVec(N , h_X , 1);
  initVec(M , h_Y , 0);
  initMat(M , N , h_A);


  float *d_A , *d_X , *d_Y;
  // allocate matrix and vector
  
  hipMalloc(&d_X, N * sizeof(float));
  hipMalloc(&d_Y, M * sizeof(float));
  hipMalloc(&d_A, M * N * sizeof(float));
    
  hipMemcpy(d_X , h_X , N * sizeof(float) ,hipMemcpyHostToDevice);
  hipMemcpy(d_A , h_A , M * N * sizeof(float) ,hipMemcpyHostToDevice);

  
  const unsigned int n_blocks = (M + threads_per_block - 1) / threads_per_block;

  std::vector<float> result_host(M);

  const unsigned int n_tests = 30;
  double best = 1e10, worst = 0, avg = 0;
  
  const unsigned int n_repeat = std::max( (unsigned int) (1) , (unsigned int) (10000 / M));
  //const unsigned int n_repeat = 1;

  // cublas constants
  float alpha = 1.f , beta = 0.;

  for (unsigned int t = 0; t < n_tests; ++t)
    {
      // type of t1: std::chrono::steady_clock::time_point
      const auto t1 = std::chrono::steady_clock::now();

      for (unsigned int rep = 0; rep < n_repeat; ++rep){
        if(useCublas){
          stat =hipblasSgemv(handle, HIPBLAS_OP_N, M, N, &alpha, d_A, M, d_X, 1, &beta, d_Y, 1);
            if (stat != HIPBLAS_STATUS_SUCCESS){
              std::cout << "CUBLAS operation failed\n";
              std::abort();
            }
        }
        else{
          dot_product<<<n_blocks, threads_per_block>>>(M , N , d_A , d_X ,d_Y);
        }
      }

      hipDeviceSynchronize();
      // measure the time by taking the difference between the time point
      // before starting and now
      const double time =
        std::chrono::duration_cast<std::chrono::duration<double>>(
          std::chrono::steady_clock::now() - t1)
          .count();

      best  = std::min(best, (double) (time / n_repeat));
      worst = std::max(worst, (double) (time / n_repeat));
      avg += time / n_repeat;
    }

  // Copy the result back to the host
  hipMemcpy(result_host.data(), d_Y, M * sizeof(float), hipMemcpyDeviceToHost);
  
  float targetResult = N * (N - 1.0) / 2.0;
  if (result_host[0] != targetResult)
    std::cout << "Error in computation, got "
              << result_host[0] << " instead of "<< targetResult
              << std::endl;

  // Free the memory on the device
  hipFree(d_A);
  hipFree(d_X);
  
  long long ops = 1ll * N * M;

  std::cout << "STREAM triad with "<< M << "rows and " << N <<" columns" 
            << " : min/avg/max: " << std::setw(11) << best << " "
            << std::setw(11) << avg / n_tests << " " << std::setw(11) << worst << " seconds or " 
            << std::setw(8) << 1e-6 * ops / best << " MUPD/s or " 
            << std::setw(8) << 1e-9 * 3 * sizeof(float) * ops / best << " GB/s" << std::endl;
}

void Task1Square(){
  
}
int main(int argc, char **argv)
{

  if (stat != HIPBLAS_STATUS_SUCCESS){
    std::cout << "CUBLAS initialization failed\n";
    std::abort();
  }

  if (argc != 2){
    std::cout << "Error, add a single argument depending on experiment";
    std::abort();
  }

  long task = static_cast<long>(std::stod(argv[1]));
  if(task == 1){
    printf("Plain CUDA:: \n");
    for(int n = 100 ; n <= 10000 ; n = (1 + n * 1.1)){
      n = (n + 7) / 8 * 8;
      benchmark_triad(n , n , 10000 , 0);
    }
    printf("CUBLAS :: \n");
    for(int n = 100 ; n <= 10000 ; n = (1 + n * 1.1)){
      n = (n + 7) / 8 * 8;
      benchmark_triad(n , n , 10000 , 1);
    }
  }

  hipblasDestroy(handle);
  
  /*
  if(task == 2){
    for(int n = 100 ; n <= 10000 ; n = (1 + n * 1.1)){
      n = (n + 7) / 8 * 8;
      benchmark_triad(n , n , 10000 , 1);
    }
  }
  if(task == 3){
    for(int m = 100 ; m <= 10000 ; m = (1 + m * 1.1)){
      m = (n + 7) / 8 * 8;
      benchmark_triad(n , n , 10000 , 1);
    }
  }*/

  
  /*
  arguments:
  m_min
  m_max
  n_min
  n_max
  repeat
  */
  /*
  

  
  long m_max = static_cast<long>(std::stod(argv[2]));
  long n_min = static_cast<long>(std::stod(argv[3]));
  long n_max = static_cast<long>(std::stod(argv[4]));
  long repeat = static_cast<long>(std::stod(argv[5]));


  for(long m = m_min ; m <= m_max ; m = (1 + m * 1.1)){
    m = (m + 7) / 8 * 8;
    for (long n = n_min; n <= n_max; n = (1 + n * 1.1)){
        // round up to nearest multiple of 8
        n = (n + 7) / 8 * 8;
        std::cout<<n<<' '<<m<<'\n';
        benchmark_triad(m , n, repeat);
    }
  }*/

  return 0;
}
